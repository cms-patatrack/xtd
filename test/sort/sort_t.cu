
#include <algorithm>
#include <numeric>
#include <ranges>
#include <vector>

#define CATCH_CONFIG_MAIN
#include <catch.hpp>

#include "algorithm.h"

#include "common/cuda_check.h"
#include <hip/hip_runtime.h>

TEST_CASE("sortCUDA", "[sort]") {
  int deviceCount;
  hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);

  if (cudaStatus != hipSuccess || deviceCount == 0) {
    std::cout << "No NVIDIA GPUs found, the test will be skipped." << std::endl;
    exit(EXIT_SUCCESS);
  }
  CUDA_CHECK(hipSetDevice(0));

  const int N = 100;
  std::random_device rd;
  std::mt19937 rng(rd());

  std::vector<int> values(N);
  std::iota(values.begin(), values.end(), 0);
  std::shuffle(values.begin(), values.end(), rng);
  xtd::sort(values.begin(), values.end());

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  int* d_values;
  CUDA_CHECK(hipMallocAsync(&d_values, N * sizeof(int), stream));
  CUDA_CHECK(hipMemcpyAsync(d_values, values.data(), N * sizeof(int), hipMemcpyHostToDevice, stream));
  xtd::sort(d_values, d_values + N);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipMemcpyAsync(values.data(), d_values, N * sizeof(int), hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipStreamSynchronize(stream));

  REQUIRE(std::ranges::equal(values, std::views::iota(N, 0)));

  CUDA_CHECK(hipFreeAsync(d_values, stream));
  CUDA_CHECK(hipStreamDestroy(stream));
}
