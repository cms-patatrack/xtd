#include "hip/hip_runtime.h"

#include <algorithm>
#include <execution>
#include <numeric>
#include <ranges>
#include <vector>

#define CATCH_CONFIG_MAIN
#include <catch.hpp>

#include "algorithm.h"

#include "common/cuda_check.h"
#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>

TEST_CASE("max_elementCUDA", "[max_element]") {
  const int N = 100;
  std::random_device rd;
  std::mt19937 rng(rd());

  std::vector<int> values(N);
  std::iota(values.begin(), values.end(), 0);
  std::shuffle(values.begin(), values.end(), rng);

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  int* d_values;
  CUDA_CHECK(hipMallocAsync(&d_values, N * sizeof(int), stream));
  CUDA_CHECK(hipMemcpyAsync(d_values, values.data(), N * sizeof(int), hipMemcpyHostToDevice, stream));

  SECTION("Default comparison") {
    auto max_iter = xtd::max_element(d_values, d_values + N);
	int max;
	thrust::copy(thrust::device, d_values, d_values + 1, &max);
    REQUIRE(max == N - 1);
  }

  SECTION("Greater comparison") {
    auto max_iter = xtd::max_element(d_values, d_values + N, std::greater<int>());
	int max;
	thrust::copy(thrust::device, d_values, d_values + 1, &max);
    REQUIRE(max == 0);
  }
}
