#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <cmath>
#include <iostream>
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_MAIN
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// mpfr::real headers
#include <real.hpp>

// xtd headers
#include "xtd/math/acosh.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"
#include "common/math_inputs.h"

constexpr int ulps_float = 4;
constexpr int ulps_double = 3;

TEST_CASE("xtd::acosh", "[acosh][cuda]") {
  std::vector<double> values = generate_input_values();

  int deviceCount;
  hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);

  if (cudaStatus != hipSuccess || deviceCount == 0) {
    std::cout << "No NVIDIA GPUs found, the test will be skipped.\n\n";
    exit(EXIT_SUCCESS);
  }

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    std::string section = "CUDA GPU "s + std::to_string(device) + ": "s + properties.name;
    SECTION(section) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::acosh(float)") {
        test<float, float, xtd::acosh, mpfr::acosh>(queue, values, ulps_float);
      }

      SECTION("double xtd::acosh(double)") {
        test<double, double, xtd::acosh, mpfr::acosh>(queue, values, ulps_double);
      }

      SECTION("double xtd::acosh(int)") {
        test<double, int, xtd::acosh, mpfr::acosh>(queue, values, ulps_double);
      }

      SECTION("float xtd::acoshf(float)") {
        test_f<float, float, xtd::acoshf, mpfr::acosh>(queue, values, ulps_float);
      }

      SECTION("float xtd::acoshf(double)") {
        test_f<float, double, xtd::acoshf, mpfr::acosh>(queue, values, ulps_float);
      }

      SECTION("float xtd::acoshf(int)") {
        test_f<float, int, xtd::acoshf, mpfr::acosh>(queue, values, ulps_float);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
