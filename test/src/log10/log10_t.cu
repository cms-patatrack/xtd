#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// mpfr::real headers
#include <real.hpp>

// xtd headers
#include "xtd/math/log10.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"
#include "common/math_inputs.h"

constexpr int ulps_float = 2;
constexpr int ulps_double = 1;

TEST_CASE("xtd::log10", "[log10][cuda]") {
  std::vector<double> values = generate_input_values();

  int deviceCount;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount));

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    std::string section = "CUDA GPU "s + std::to_string(device) + ": "s + properties.name;
    SECTION(section) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::log10(float)") {
        test<float, float, xtd::log10, mpfr::log10>(queue, values, ulps_float);
      }

      SECTION("double xtd::log10(double)") {
        test<double, double, xtd::log10, mpfr::log10>(queue, values, ulps_double);
      }

      SECTION("double xtd::log10(int)") {
        test<double, int, xtd::log10, mpfr::log10>(queue, values, ulps_double);
      }

      SECTION("float xtd::log10f(float)") {
        test_f<float, float, xtd::log10f, mpfr::log10>(queue, values, ulps_float);
      }

      SECTION("float xtd::log10f(double)") {
        test_f<float, double, xtd::log10f, mpfr::log10>(queue, values, ulps_float);
      }

      SECTION("float xtd::log10f(int)") {
        test_f<float, int, xtd::log10f, mpfr::log10>(queue, values, ulps_float);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
