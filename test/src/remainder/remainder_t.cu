#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// mpfr::real headers
#include <real.hpp>

// xtd headers
#include "xtd/math/remainder.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"
#include "common/math_inputs.h"

constexpr int ulps_float = 0;
constexpr int ulps_double = 0;

constexpr auto ref_remainder = [](mpfr_double y, mpfr_double x) -> mpfr_double { return mpfr::remainder(y, x); };
constexpr auto ref_remainderf = [](mpfr_single y, mpfr_single x) -> mpfr_single { return mpfr::remainder(y, x); };

TEST_CASE("xtd::remainder", "[remainder][cuda]") {
  std::vector<double> values = generate_input_values();

  int deviceCount;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount));

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    DYNAMIC_SECTION("CUDA device " << device << ": " << properties.name) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::remainder(float, float)") {
        test_2<float, float, xtd::remainder, ref_remainder>(queue, values, ulps_float);
      }

      SECTION("double xtd::remainder(double, double)") {
        test_2<double, double, xtd::remainder, ref_remainder>(queue, values, ulps_double);
      }

      SECTION("double xtd::remainder(int, int)") {
        test_2<double, int, xtd::remainder, ref_remainder>(queue, values, ulps_double);
      }

      SECTION("float xtd::remainderf(float, float)") {
        test_2f<float, float, xtd::remainderf, ref_remainderf>(queue, values, ulps_float);
      }

      SECTION("float xtd::remainderf(double, double)") {
        test_2f<float, double, xtd::remainderf, ref_remainderf>(queue, values, ulps_float);
      }

      SECTION("float xtd::remainderf(int, int)") {
        test_2f<float, int, xtd::remainderf, ref_remainderf>(queue, values, ulps_float);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
