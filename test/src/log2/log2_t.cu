#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// mpfr::real headers
#include <real.hpp>

// xtd headers
#include "xtd/math/log2.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"
#include "common/math_inputs.h"

constexpr int ulps_float = 1;
constexpr int ulps_double = 1;

TEST_CASE("xtd::log2", "[log2][cuda]") {
  std::vector<double> values = generate_input_values();

  int deviceCount;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount));

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    std::string section = "CUDA GPU "s + std::to_string(device) + ": "s + properties.name;
    SECTION(section) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::log2(float)") {
        test<float, float, xtd::log2, mpfr::log2>(queue, values, ulps_float);
      }

      SECTION("double xtd::log2(double)") {
        test<double, double, xtd::log2, mpfr::log2>(queue, values, ulps_double);
      }

      SECTION("double xtd::log2(int)") {
        test<double, int, xtd::log2, mpfr::log2>(queue, values, ulps_double);
      }

      SECTION("float xtd::log2f(float)") {
        test_f<float, float, xtd::log2f, mpfr::log2>(queue, values, ulps_float);
      }

      SECTION("float xtd::log2f(double)") {
        test_f<float, double, xtd::log2f, mpfr::log2>(queue, values, ulps_float);
      }

      SECTION("float xtd::log2f(int)") {
        test_f<float, int, xtd::log2f, mpfr::log2>(queue, values, ulps_float);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
