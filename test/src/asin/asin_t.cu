#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// mpfr::real headers
#include <real.hpp>

// xtd headers
#include "xtd/math/asin.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"
#include "common/math_inputs.h"

constexpr int ulps_float = 2;
constexpr int ulps_double = 2;

TEST_CASE("xtd::asin", "[asin][cuda]") {
  std::vector<double> values = generate_input_values();

  int deviceCount;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount));

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    DYNAMIC_SECTION("CUDA device " << device << ": " << properties.name) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::asin(float)") {
        test<float, float, xtd::asin, mpfr::asin>(queue, values, ulps_float);
      }

      SECTION("double xtd::asin(double)") {
        test<double, double, xtd::asin, mpfr::asin>(queue, values, ulps_double);
      }

      SECTION("double xtd::asin(int)") {
        test<double, int, xtd::asin, mpfr::asin>(queue, values, ulps_double);
      }

      SECTION("float xtd::asinf(float)") {
        test_f<float, float, xtd::asinf, mpfr::asin>(queue, values, ulps_float);
      }

      SECTION("float xtd::asinf(double)") {
        test_f<float, double, xtd::asinf, mpfr::asin>(queue, values, ulps_float);
      }

      SECTION("float xtd::asinf(int)") {
        test_f<float, int, xtd::asinf, mpfr::asin>(queue, values, ulps_float);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
