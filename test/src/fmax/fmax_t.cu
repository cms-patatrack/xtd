#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// mpfr::real headers
#include <real.hpp>

// xtd headers
#include "xtd/math/fmax.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"
#include "common/math_inputs.h"

constexpr int ulps_float = 0;
constexpr int ulps_double = 0;

constexpr auto ref_fmax = [](mpfr_double y, mpfr_double x) -> mpfr_double { return mpfr::fmax(y, x); };
constexpr auto ref_fmaxf = [](mpfr_single y, mpfr_single x) -> mpfr_single { return mpfr::fmax(y, x); };

TEST_CASE("xtd::fmax", "[fmax][cuda]") {
  std::vector<double> values = generate_input_values();

  int deviceCount;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount));

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    DYNAMIC_SECTION("CUDA device " << device << ": " << properties.name) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::fmax(float, float)") {
        test_2<float, float, xtd::fmax, ref_fmax>(queue, values, ulps_float);
      }

      SECTION("double xtd::fmax(double, double)") {
        test_2<double, double, xtd::fmax, ref_fmax>(queue, values, ulps_double);
      }

      SECTION("double xtd::fmax(int, int)") {
        test_2<double, int, xtd::fmax, ref_fmax>(queue, values, ulps_double);
      }

      SECTION("float xtd::fmaxf(float, float)") {
        test_2f<float, float, xtd::fmaxf, ref_fmaxf>(queue, values, ulps_float);
      }

      SECTION("float xtd::fmaxf(double, double)") {
        test_2f<float, double, xtd::fmaxf, ref_fmaxf>(queue, values, ulps_float);
      }

      SECTION("float xtd::fmaxf(int, int)") {
        test_2f<float, int, xtd::fmaxf, ref_fmaxf>(queue, values, ulps_float);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
