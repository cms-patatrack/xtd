#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <cmath>
#include <iostream>
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// mpfr::real headers
#include <real.hpp>

// xtd headers
#include "xtd/math/fmod.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"
#include "common/math_inputs.h"

constexpr int ulps_float = 0;
constexpr int ulps_double = 0;

constexpr auto ref_fmod = [](mpfr_double y, mpfr_double x) -> mpfr_double { return mpfr::fmod(y, x); };
constexpr auto ref_fmodf = [](mpfr_single y, mpfr_single x) -> mpfr_single { return mpfr::fmod(y, x); };

TEST_CASE("xtd::fmod", "[fmod][cuda]") {
  std::vector<double> values = generate_input_values();

  int deviceCount;
  hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);

  if (cudaStatus != hipSuccess || deviceCount == 0) {
    std::cout << "No NVIDIA GPUs found, the test will be skipped.\n\n";
    exit(EXIT_SUCCESS);
  }

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    std::string section = "CUDA GPU "s + std::to_string(device) + ": "s + properties.name;
    SECTION(section) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::fmod(float, float)") {
        test_2<float, float, xtd::fmod, ref_fmod>(queue, values, ulps_float);
      }

      SECTION("double xtd::fmod(double, double)") {
        test_2<double, double, xtd::fmod, ref_fmod>(queue, values, ulps_double);
      }

      SECTION("double xtd::fmod(int, int)") {
        test_2<double, int, xtd::fmod, ref_fmod>(queue, values, ulps_double);
      }

      SECTION("float xtd::fmodf(float, float)") {
        test_2f<float, float, xtd::fmodf, ref_fmodf>(queue, values, ulps_float);
      }

      SECTION("float xtd::fmodf(double, double)") {
        test_2f<float, double, xtd::fmodf, ref_fmodf>(queue, values, ulps_float);
      }

      SECTION("float xtd::fmodf(int, int)") {
        test_2f<float, int, xtd::fmodf, ref_fmodf>(queue, values, ulps_float);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
