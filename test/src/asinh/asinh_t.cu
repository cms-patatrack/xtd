#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// mpfr::real headers
#include <real.hpp>

// xtd headers
#include "xtd/math/asinh.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"
#include "common/math_inputs.h"

constexpr int ulps_float = 3;
constexpr int ulps_double = 3;

TEST_CASE("xtd::asinh", "[asinh][cuda]") {
  std::vector<double> values = generate_input_values();

  int deviceCount;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount));

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    DYNAMIC_SECTION("CUDA device " << device << ": " << properties.name) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::asinh(float)") {
        test<float, float, xtd::asinh, mpfr::asinh>(queue, values, ulps_float);
      }

      SECTION("double xtd::asinh(double)") {
        test<double, double, xtd::asinh, mpfr::asinh>(queue, values, ulps_double);
      }

      SECTION("double xtd::asinh(int)") {
        test<double, int, xtd::asinh, mpfr::asinh>(queue, values, ulps_double);
      }

      SECTION("float xtd::asinhf(float)") {
        test_f<float, float, xtd::asinhf, mpfr::asinh>(queue, values, ulps_float);
      }

      SECTION("float xtd::asinhf(double)") {
        test_f<float, double, xtd::asinhf, mpfr::asinh>(queue, values, ulps_float);
      }

      SECTION("float xtd::asinhf(int)") {
        test_f<float, int, xtd::asinhf, mpfr::asinh>(queue, values, ulps_float);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
