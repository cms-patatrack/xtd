#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// mpfr::real headers
#include <real.hpp>

// xtd headers
#include "xtd/math/ceil.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"
#include "common/math_inputs.h"

constexpr int ulps_float = 0;
constexpr int ulps_double = 0;

TEST_CASE("xtd::ceil", "[ceil][cuda]") {
  std::vector<double> values = generate_input_values();

  int deviceCount;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount));

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    DYNAMIC_SECTION("CUDA device " << device << ": " << properties.name) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::ceil(float)") {
        test<float, float, xtd::ceil, mpfr::ceil>(queue, values, ulps_float);
      }

      SECTION("double xtd::ceil(double)") {
        test<double, double, xtd::ceil, mpfr::ceil>(queue, values, ulps_double);
      }

      SECTION("double xtd::ceil(int)") {
        test<double, int, xtd::ceil, mpfr::ceil>(queue, values, ulps_double);
      }

      SECTION("float xtd::ceilf(float)") {
        test_f<float, float, xtd::ceilf, mpfr::ceil>(queue, values, ulps_float);
      }

      SECTION("float xtd::ceilf(double)") {
        test_f<float, double, xtd::ceilf, mpfr::ceil>(queue, values, ulps_float);
      }

      SECTION("float xtd::ceilf(int)") {
        test_f<float, int, xtd::ceilf, mpfr::ceil>(queue, values, ulps_float);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
