#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// mpfr::real headers
#include <real.hpp>

// xtd headers
#include "xtd/math/abs.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"
#include "common/math_inputs.h"

constexpr int ulps_float = 0;
constexpr int ulps_double = 0;

TEST_CASE("xtd::abs", "[abs][cuda]") {
  std::vector<double> values = generate_input_values();

  int deviceCount;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount));

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    DYNAMIC_SECTION("CUDA device " << device << ": " << properties.name) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::abs(float)") {
        test<float, float, xtd::abs, mpfr::fabs>(queue, values, ulps_float);
      }

      SECTION("double xtd::abs(double)") {
        test<double, double, xtd::abs, mpfr::fabs>(queue, values, ulps_double);
      }

      SECTION("int xtd::abs(int)") {
        test_i<int, xtd::abs, std::abs>(queue, values);
      }

      SECTION("long long xtd::abs(long long)") {
        test_i<long long, xtd::abs, std::abs>(queue, values);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
