#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <cmath>
#include <iostream>
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_MAIN
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// mpfr::real headers
#include <real.hpp>

// xtd headers
#include "xtd/math/floor.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"
#include "common/math_inputs.h"

constexpr int ulps_float = 0;
constexpr int ulps_double = 0;

TEST_CASE("xtd::floor", "[floor][cuda]") {
  std::vector<double> values = generate_input_values();

  int deviceCount;
  hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);

  if (cudaStatus != hipSuccess || deviceCount == 0) {
    std::cout << "No NVIDIA GPUs found, the test will be skipped.\n\n";
    exit(EXIT_SUCCESS);
  }

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    std::string section = "CUDA GPU "s + std::to_string(device) + ": "s + properties.name;
    SECTION(section) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::floor(float)") {
        test<float, float, xtd::floor, mpfr::floor>(queue, values, ulps_float);
      }

      SECTION("double xtd::floor(double)") {
        test<double, double, xtd::floor, mpfr::floor>(queue, values, ulps_double);
      }

      SECTION("double xtd::floor(int)") {
        test<double, int, xtd::floor, mpfr::floor>(queue, values, ulps_double);
      }

      SECTION("float xtd::floorf(float)") {
        test_f<float, float, xtd::floorf, mpfr::floor>(queue, values, ulps_float);
      }

      SECTION("float xtd::floorf(double)") {
        test_f<float, double, xtd::floorf, mpfr::floor>(queue, values, ulps_float);
      }

      SECTION("float xtd::floorf(int)") {
        test_f<float, int, xtd::floorf, mpfr::floor>(queue, values, ulps_float);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
