#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <cmath>
#include <iostream>
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// mpfr::real headers
#include <real.hpp>

// xtd headers
#include "xtd/math/sqrt.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"
#include "common/math_inputs.h"

constexpr int ulps_float = 0;
constexpr int ulps_double = 0;

TEST_CASE("xtd::sqrt", "[sqrt][cuda]") {
  std::vector<double> values = generate_input_values();

  int deviceCount;
  hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);

  if (cudaStatus != hipSuccess || deviceCount == 0) {
    std::cout << "No NVIDIA GPUs found, the test will be skipped.\n\n";
    exit(EXIT_SUCCESS);
  }

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    std::string section = "CUDA GPU "s + std::to_string(device) + ": "s + properties.name;
    SECTION(section) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::sqrt(float)") {
        test<float, float, xtd::sqrt, mpfr::sqrt>(queue, values, ulps_float);
      }

      SECTION("double xtd::sqrt(double)") {
        test<double, double, xtd::sqrt, mpfr::sqrt>(queue, values, ulps_double);
      }

      SECTION("double xtd::sqrt(int)") {
        test<double, int, xtd::sqrt, mpfr::sqrt>(queue, values, ulps_double);
      }

      SECTION("float xtd::sqrtf(float)") {
        test_f<float, float, xtd::sqrtf, mpfr::sqrt>(queue, values, ulps_float);
      }

      SECTION("float xtd::sqrtf(double)") {
        test_f<float, double, xtd::sqrtf, mpfr::sqrt>(queue, values, ulps_float);
      }

      SECTION("float xtd::sqrtf(int)") {
        test_f<float, int, xtd::sqrtf, mpfr::sqrt>(queue, values, ulps_float);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
