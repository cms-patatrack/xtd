#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// mpfr::real headers
#include <real.hpp>

// xtd headers
#include "xtd/math/cbrt.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"
#include "common/math_inputs.h"

constexpr int ulps_float = 1;
constexpr int ulps_double = 1;

TEST_CASE("xtd::cbrt", "[cbrt][cuda]") {
  std::vector<double> values = generate_input_values();

  int deviceCount;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount));

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    std::string section = "CUDA GPU "s + std::to_string(device) + ": "s + properties.name;
    SECTION(section) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::cbrt(float)") {
        test<float, float, xtd::cbrt, mpfr::cbrt>(queue, values, ulps_float);
      }

      SECTION("double xtd::cbrt(double)") {
        test<double, double, xtd::cbrt, mpfr::cbrt>(queue, values, ulps_double);
      }

      SECTION("double xtd::cbrt(int)") {
        test<double, int, xtd::cbrt, mpfr::cbrt>(queue, values, ulps_double);
      }

      SECTION("float xtd::cbrtf(float)") {
        test_f<float, float, xtd::cbrtf, mpfr::cbrt>(queue, values, ulps_float);
      }

      SECTION("float xtd::cbrtf(double)") {
        test_f<float, double, xtd::cbrtf, mpfr::cbrt>(queue, values, ulps_float);
      }

      SECTION("float xtd::cbrtf(int)") {
        test_f<float, int, xtd::cbrtf, mpfr::cbrt>(queue, values, ulps_float);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
