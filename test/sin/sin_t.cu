#include "hip/hip_runtime.h"
/*
 * Copyright 2025 European Organization for Nuclear Research (CERN)
 * Authors: Andrea Bocci <andrea.bocci@cern.ch>, Aurora Perego <aurora.perego@cern.ch>
 * SPDX-License-Identifier: MPL-2.0
 */

// C++ standard headers
#include <cmath>
#include <iostream>
#include <string>
#include <vector>
using namespace std::literals;

// Catch2 headers
#define CATCH_CONFIG_MAIN
#define CATCH_CONFIG_NO_POSIX_SIGNALS
#include <catch.hpp>

// CUDA headers
#include <hip/hip_runtime.h>

// xtd headers
#include "math/sin.h"

// test headers
#include "common/cuda_check.h"
#include "common/cuda_test.h"

TEST_CASE("xtd::sin", "[sin][cuda]") {
  int deviceCount;
  hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);

  if (cudaStatus != hipSuccess || deviceCount == 0) {
    std::cout << "No NVIDIA GPUs found, the test will be skipped." << std::endl;
    exit(EXIT_SUCCESS);
  }

  std::vector<double> values{-1., 0., M_PI / 2, M_PI, 42.};

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t properties;
    CUDA_CHECK(hipGetDeviceProperties(&properties, device));
    std::string section = "CUDA GPU "s + std::to_string(device) + ": "s + properties.name;
    SECTION(section) {
      // set the current GPU
      CUDA_CHECK(hipSetDevice(device));

      // create a CUDA stream for all the asynchronous operations on this GPU
      hipStream_t queue;
      CUDA_CHECK(hipStreamCreate(&queue));

      SECTION("float xtd::sin(float)") {
        test<float, float, xtd::sin, std::sin>(queue, values);
      }

      SECTION("double xtd::sin(double)") {
        test<double, double, xtd::sin, std::sin>(queue, values);
      }

      SECTION("double xtd::sin(int)") {
        test<double, int, xtd::sin, std::sin>(queue, values);
      }

      // Note: GCC prior to v14.1 and clang prior to v19.1 do not provide std::sinf().
      // As a workarund, use C sinf().

      SECTION("float xtd::sinf(float)") {
        test_f<float, float, xtd::sinf, ::sinf>(queue, values);
      }

      SECTION("float xtd::sinf(double)") {
        test_f<float, double, xtd::sinf, ::sinf>(queue, values);
      }

      SECTION("float xtd::sinf(int)") {
        test_f<float, int, xtd::sinf, ::sinf>(queue, values);
      }

      CUDA_CHECK(hipStreamDestroy(queue));
    }
  }
}
